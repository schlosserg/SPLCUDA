#include "hip/hip_runtime.h"
﻿// SPL.cpp: Definiert den Einstiegspunkt für die Anwendung.
//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <spl/vector3.hh>

//using namespace std;

//__global__ void cuda_hello();

__global__ void cuda_hello() {
	printf("Hello World from GPU!\n");
}

int main()
{

	SPLVector3<SPLieee32> vec, vec1, vec2;

	vec1 = SPLVector3<SPLieee32>(1, 1, 1);
	vec2 = SPLVector3<SPLieee32>(2, 2, 2);

	vec = vec1 + vec2;
	vec.print();

	cuda_hello<<<1, 1>>>();

	return 0;
}
